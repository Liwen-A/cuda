#include "hip/hip_runtime.h"
#define ARMA_ALLOW_FAKE_GCC
#include <algorithm>
#include <armadillo>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include <memory>

#include "gpu_func.h"
#include "util.cuh"

__global__ void Warmup() {}

void DWarmup() { Warmup<<<1, 1>>>(); }

/**
 * DeviceAllocator and DeviceMatrix
 */

DeviceAllocator::DeviceAllocator(nn_real *cpu_data, int n) {
  assert(n >= 0);
  assert(cpu_data != nullptr);
  nbytes = n * sizeof(nn_real);
  hipMalloc(&data, nbytes);
  hipMemcpy(data, cpu_data, nbytes, hipMemcpyHostToDevice);
}

DeviceAllocator::DeviceAllocator(int n) {
  assert(n >= 0);
  nbytes = n * sizeof(nn_real);
  hipMalloc(&data, nbytes);
}

DeviceAllocator::~DeviceAllocator() {
  if (data != nullptr)
    hipFree(data);
}

int DeviceAllocator::total_bytes() { return nbytes; }

nn_real *DeviceAllocator::memptr() { return data; }

void DeviceAllocator::to_cpu(nn_real *cpu_data) {
  assert(data != nullptr && cpu_data != nullptr);
  hipMemcpy(cpu_data, data, nbytes, hipMemcpyDeviceToHost);
}

DeviceMatrix::DeviceMatrix(int n_rows, int n_cols) {
  assert(n_rows >= 0 && n_cols >= 0);
  this->allocator = std::make_shared<DeviceAllocator>(n_rows * n_cols);
  this->data = this->allocator->memptr();
  this->n_rows = n_rows;
  this->n_cols = n_cols;
}

DeviceMatrix::DeviceMatrix(arma::Mat<nn_real> &cpu_mat) {
  this->allocator = std::make_shared<DeviceAllocator>(
      cpu_mat.memptr(), cpu_mat.n_rows * cpu_mat.n_cols);
  this->data = this->allocator->memptr();
  this->n_rows = cpu_mat.n_rows;
  this->n_cols = cpu_mat.n_cols;
}

int DeviceMatrix::total_bytes() { return allocator->total_bytes(); }

nn_real *DeviceMatrix::memptr() { return data; }

void DeviceMatrix::to_cpu(arma::Mat<nn_real> &cpu_mat) {
  allocator->to_cpu(cpu_mat.memptr());
}

__device__ nn_real &DeviceMatrix::operator()(int row, int col, bool transpose) {
  assert(data != nullptr && row >= 0 && row < n_rows && col >= 0 &&
         col < n_cols);
  return transpose ? data[row * n_cols + col] : data[col * n_rows + row];
}
// +-*=+-*=+-*=+-*=+-*=+-*=+-*=+-*=+*-=+-*=+*-=+-*=+-*=+-*=+-*=+-*= //
//                           GEMM kernels                           //
// +-*=+-*=+-*=+-*=+-*=+-*=+-*=+-*=+*-=+-*=+*-=+-*=+-*=+-*=+-*=+-*= //

__global__ void BasicMatMulColumnMajor(DeviceMatrix A, DeviceMatrix B,
                                       DeviceMatrix C, nn_real alpha,
                                       nn_real beta) {
  // TODO: Implement this kernel
  int id = blockDim.x * blockIdx.x + threadIdx.x; 
  if (id < C.n_cols * C.n_rows){
    int i = id % C.n_rows;
    int j = id / C.n_rows;
    float cum_sum = 0.0;
    for (int k = 0; k < A.n_cols; k++){
      cum_sum += A(i,k) * B(k,j) ;
    }
    C(i,j) = alpha * cum_sum + beta * C(i,j);
  }
}

void basicGEMMColumnMajor(DeviceMatrix A, DeviceMatrix B, DeviceMatrix C,
                          nn_real alpha, nn_real beta) {
  // TODO: Implement this kernel wrapper
  // Remember that column major means that consecutive threads compute
  // consecutive elements in a column of the output matrix
  int numThread = 32*32;
  int numBlock = C.n_cols * C.n_rows / numThread+1;
  dim3 blockPerGrid(numBlock);
  dim3 threadPerBlock(numThread);
  BasicMatMulColumnMajor<<<blockPerGrid,threadPerBlock>>>(A,B,C,alpha,beta);
  check_launch("basicGEMMColumnMajor");
}

__global__ void BasicMatMulRowMajor(DeviceMatrix A, DeviceMatrix B,
                                    DeviceMatrix C, nn_real alpha,
                                    nn_real beta) {
  // TODO: Implement this kernel
  int id = blockDim.x * blockIdx.x + threadIdx.x; 
  if (id < C.n_cols * C.n_rows){
    int i = id / C.n_cols;
    int j = id % C.n_cols;
    float cum_sum = 0.0;
    for (int k = 0; k < A.n_cols; k++){
      cum_sum += A(i,k) * B(k,j) ;
    }
    C(i,j) = alpha * cum_sum + beta * C(i,j);
  }
}

void basicGEMMRowMajor(DeviceMatrix A, DeviceMatrix B, DeviceMatrix C,
                       nn_real alpha, nn_real beta) {
  // TODO: Implement this kernel wrapper
  // Remember that row major means that consecutive threads compute
  // consecutive elements in a row of the output matrix
  int numThread = 32*32;
  int numBlock = C.n_cols * C.n_rows / numThread+1;
  dim3 blockPerGrid(numBlock);
  dim3 threadPerBlock(numThread);
  BasicMatMulColumnMajor<<<blockPerGrid,threadPerBlock>>>(A,B,C,alpha,beta);
  check_launch("basicGEMMRowMajor");
}

template <int blockSizeX, int blockSizeY>
__global__ void SharedMemoryMatMul(DeviceMatrix A, DeviceMatrix B,
                                   DeviceMatrix C, nn_real alpha,
                                   nn_real beta) {

  // TODO: Implement this kernel
    float CValue = 0;

    int Row = blockIdx.y*blockSizeY + threadIdx.y;
    int Col = blockIdx.x*blockSizeX+ threadIdx.x;

    __shared__ float As[blockSizeX][blockSizeY];
    __shared__ float Bs[blockSizeX][blockSizeY];
    for (int k = 0; k < (blockSizeX + A.n_cols - 1)/blockSizeX; k++) {

      if (k*blockSizeX + threadIdx.x < A.n_cols && Row < A.n_rows)
          As[threadIdx.y][threadIdx.x] = A(Row , k*blockSizeX+threadIdx.x); 
      else
          As[threadIdx.y][threadIdx.x] = 0.0;

      if (k*blockSizeY + threadIdx.y < B.n_rows && Col < B.n_cols)
          Bs[threadIdx.y][threadIdx.x] =  B(k*blockSizeY+threadIdx.y, Col);
      else
          Bs[threadIdx.y][threadIdx.x] = 0.0;

      __syncthreads();

      for (int n = 0; n < blockSizeY; ++n)
          CValue += alpha*As[threadIdx.y][n] * Bs[n][threadIdx.x];

      __syncthreads();
    }

    if (Row < C.n_rows && Col < C.n_cols){
      C(Row,Col) = CValue + beta * C(Row,Col);
    }
}

void sharedMemoryGEMM(DeviceMatrix A, DeviceMatrix B, DeviceMatrix C,
                      nn_real alpha, nn_real beta) {
  // TODO: Implement this wrapper
  int blockX = C.n_cols/32+1;
  int blockY = C.n_rows/32+1;
  dim3 blockPerGrid(blockX,blockY);
  dim3 threadPerBlock(32,32);
  SharedMemoryMatMul<32,32><<<blockPerGrid,threadPerBlock>>>(A,B,C,alpha,beta);
  check_launch("sharedMemoryGEMM");
}

// 32x32 Hierarchical Tiling
// num_thread: number of threads per block
// blockItemsM: number of rows of A in each submatrix of A
// blockItemsN: number of columns of B in each submatrix of B
// blockItemsK: number of columns in submatrix of A and rows in submatrix of B
template <int num_thread, int blockItemsM, int blockItemsN, int blockItemsK>
__global__ void TiledMatMul(DeviceMatrix A, bool transa, DeviceMatrix B,
                            bool transb, DeviceMatrix C, nn_real alpha,
                            nn_real beta) {
  // TODO: Implement this kernel
}

// wrapper for MatMulTile_32_32
void tiledGEMM(DeviceMatrix A, DeviceMatrix B, DeviceMatrix C, nn_real alpha,
               nn_real beta) {
  assert((A.n_cols) == (B.n_rows));
  assert(C.n_rows == (A.n_rows) && C.n_cols == (B.n_cols));

  constexpr int block_m = 32;
  constexpr int block_n = 32;
  constexpr int block_k = 32;
  constexpr int num_thread = 128;
  dim3 grid((C.n_rows + block_m - 1) / block_m,
            (C.n_cols + block_n - 1) / block_n);
  TiledMatMul<num_thread, block_m, block_n, block_k>
      <<<grid, num_thread>>>(A, false, B, false, C, alpha, beta);

  check_launch("tiledGEMM");
}
